#include "hip/hip_runtime.h"
#include"external_libs"





__global__ void seed_curand_xor( int size, int seed, hiprandState* states) {
	DIMS_1D(id);
	BOUNDS_1D(size);

	hiprand_init(seed, id, 0, &states[id]);
}




namespace on {
	
	On_Structure Random {

		hiprandState* Initialize::curand_xor(int size, int seed) {
			
			hiprandState* states;
			hipMalloc((void**)&states, size * sizeof(hiprandState));
		
			sk::configure::kernel_1d(size);
			seed_curand_xor<<<LAUNCH>>>(size, seed, states);
			//SYNC_KERNEL(seed_curand_xor);

			return states;
		}

	}



}